#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include "define.cuh"

extern "C" hipError_t InitCuda(const int w, const int h, unsigned char **dev_bitmap);
extern "C" hipError_t CalculateCuda(const int w, const int h, unsigned char *dev_bitmap,unsigned char *host_bitmap);


__global__ void Raykernel(const int w, const int h, unsigned char *dev_bitmap)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	if (x < w && y < h) {
		int offset = x + y * w;
		dev_bitmap[offset * 4] = 255;
		dev_bitmap[offset * 4 + 1] = 0;
		dev_bitmap[offset * 4 + 2] = 0;
		dev_bitmap[offset * 4 + 3] = 255;
	}
}

hipError_t InitCuda(const int w, const int h, unsigned char **dev_bitmap)
{
	const int imageSize = w*h * 4;
	hipError_t cudaStatus = hipMalloc(dev_bitmap, imageSize * sizeof(int));
	return cudaStatus;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t CalculateCuda(const int w, const int h, unsigned char *dev_bitmap, unsigned char *host_bitmap)
{
	const int imageSize = w*h * 4;
	dim3 blocks((w + DIM - 1) / DIM, (h + DIM - 1) / DIM);
	dim3 threads(DIM, DIM);
	// Launch a kernel on the GPU with one thread for each element.
	Raykernel << <blocks, threads >> > (w, h, dev_bitmap);

	hipError_t cudaStatus = hipMemcpy(host_bitmap, dev_bitmap, imageSize, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
	}
	return cudaStatus;
}
