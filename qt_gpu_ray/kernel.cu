#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include "define.cuh"
#include "Sphere.cuh"
#include "shaderec.cuh"

extern "C" hipError_t InitCuda(const int w, const int h, unsigned char **dev_bitmap);
extern "C" hipError_t CalculateCuda(const int w, const int h, unsigned char *dev_bitmap,unsigned char *host_bitmap);


__global__ void Raykernel(const int w, const int h, unsigned char *dev_bitmap)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	if (x < w && y < h) {
		int offset = x + y * w;
		Sphere sp(Vec3(0,0,0),60);
		Ray r(Vec3(x, y, 100), Vec3(0, 0, -1));
//		ShaderRec sr;
		double tmin = 0.0001;
		if (1) {
			dev_bitmap[offset * 4] = 0;
			dev_bitmap[offset * 4 + 1] = 0;
			dev_bitmap[offset * 4 + 2] = 255;
			dev_bitmap[offset * 4 + 3] = 255;
		}
		else {
			dev_bitmap[offset * 4] = 0;
			dev_bitmap[offset * 4 + 1] = 0;
			dev_bitmap[offset * 4 + 2] = 0;
			dev_bitmap[offset * 4 + 3] = 255;
		}
	}
}

hipError_t InitCuda(const int w, const int h, unsigned char **dev_bitmap)
{
	const int imageSize = w*h * 4;
	hipError_t cudaStatus = hipMalloc(dev_bitmap, imageSize * sizeof(int));
	return cudaStatus;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t CalculateCuda(const int w, const int h, unsigned char *dev_bitmap, unsigned char *host_bitmap)
{
	const int imageSize = w*h * 4;
	dim3 blocks((w + DIM - 1) / DIM, (h + DIM - 1) / DIM);
	dim3 threads(DIM, DIM);
	// Launch a kernel on the GPU with one thread for each element.
	Raykernel << <blocks, threads >> > (w, h, dev_bitmap);

	hipError_t cudaStatus = hipMemcpy(host_bitmap, dev_bitmap, imageSize, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
	}
	return cudaStatus;
}
